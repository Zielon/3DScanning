#include "hip/hip_runtime.h"





__global__ void add(int *c, const int *a, const int *b)
{
	int i = threadIdx.x;
	c[i] = a[i] + b[i];
}